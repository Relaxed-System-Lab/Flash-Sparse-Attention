#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>   // <-- ADDED for bfloat16 support
#include <torch/extension.h>
#include <cmath>
#include <c10/cuda/CUDAException.h>
#include <c10/cuda/CUDAStream.h>
#include <c10/cuda/CUDAGuard.h>


// Define constants for the kernel.
constexpr int BLOCK_DIM_X = 128;

// __device__ functions for math, equivalent to tl.exp2 and tl.log2
__device__ inline float exp2_fast(float x) {
    return exp2f(x);
}

__device__ inline float log2_fast(float x) {
    return log2f(x);
}

#define WARP_SIZE 32

// Helper to determine padding for shared memory to avoid bank conflicts
// For int/float, typically 32 banks. If size is a multiple of 32, add 1.
#define SMEM_PAD_INT_OR_FLOAT ((WARP_SIZE % 32 == 0) ? 1 : 0)

template<int BLOCK_SIZE_D, int TOPK>
__global__ void lse_reduce_kernel_cuda(
    // Pointers
    float* lse_ptr,
    const float* m_ij_ptr,
    const float* l_ij_first_ptr,
    const float* l_ij_rest_ptr,
    const float* m_ij_last_ptr,
    const int* t_ptr,
    const int* token_index_mapping_ptr,
    // Scalars
    int start_head_id,
    int total_len,
    // Strides
    int64_t stride_lse_n,
    int64_t stride_m_ij_b, int64_t stride_m_ij_n,
    int64_t stride_l_ij_fb, int64_t stride_l_ij_fn,
    int64_t stride_l_ij_rb, int64_t stride_l_ij_rn,
    int64_t stride_tn, int64_t stride_tk,
    int64_t stride_tim_b, int64_t stride_tim_n
) {
    // const int tid = threadIdx.x; // 0 to WARP_SIZE-1
    // const int pid_q_j = blockIdx.x; // Each block handles one token
    const int tid = threadIdx.x;
    const int pid_q_j = blockIdx.x * blockDim.x + tid;

    if (pid_q_j >= total_len) {
        return;
    }

    // ===================================================================================
    // SHARED MEMORY SETUP
    // ===================================================================================
    extern __shared__ char smem_storage[]; // Dynamically sized shared memory

    // Calculate offsets for different shared memory regions
    // All sizes are per-thread * padded.
    size_t offset_t_shared = 0;
    size_t offset_real_token_index_shared = offset_t_shared + sizeof(int) * WARP_SIZE * (TOPK + SMEM_PAD_INT_OR_FLOAT);

    // Two stages for pipelined data
    size_t offset_m_ij_stage0 = offset_real_token_index_shared + sizeof(int) * WARP_SIZE * (TOPK + SMEM_PAD_INT_OR_FLOAT);
    size_t offset_m_ij_stage1 = offset_m_ij_stage0 + sizeof(float) * WARP_SIZE; // One float per thread per stage

    size_t offset_l_ij_stage0 = offset_m_ij_stage1 + sizeof(float) * WARP_SIZE;
    size_t offset_l_ij_stage1 = offset_l_ij_stage0 + sizeof(float) * WARP_SIZE;

    // BLOCK_SIZE_D elements for o_tiles. Each thread loads BLOCK_SIZE_D / WARP_SIZE.
    // Total size is BLOCK_SIZE_D * sizeof(__hip_bfloat16) * 2 stages
    // We need to ensure o_tiles_sh_stage0/1 are aligned for bf16 access.
    // Assuming BLOCK_SIZE_D is a multiple of WARP_SIZE for simplicity.
    // Padding here is for the entire BLOCK_SIZE_D if needed, not per-thread
    // size_t offset_o_tiles_stage0 = offset_l_ij_stage1 + sizeof(float) * WARP_SIZE;
    // size_t offset_o_tiles_stage1 = offset_o_tiles_stage0 + sizeof(__hip_bfloat16) * BLOCK_SIZE_D;

    // size_t offset_acc_o_scales_stage0 = offset_o_tiles_stage1 + sizeof(__hip_bfloat16) * BLOCK_SIZE_D;
    // size_t offset_acc_o_scales_stage1 = offset_acc_o_scales_stage0 + sizeof(float) * WARP_SIZE;

    // Pointers to shared memory regions
    int* t_shared = (int*)(smem_storage + offset_t_shared);
    int* real_token_index_shared = (int*)(smem_storage + offset_real_token_index_shared);

    float* m_ij_sh_stage0 = (float*)(smem_storage + offset_m_ij_stage0);
    float* m_ij_sh_stage1 = (float*)(smem_storage + offset_m_ij_stage1);
    float* l_ij_sh_stage0 = (float*)(smem_storage + offset_l_ij_stage0);
    float* l_ij_sh_stage1 = (float*)(smem_storage + offset_l_ij_stage1);
    // __hip_bfloat16* o_tiles_sh_stage0 = (__hip_bfloat16*)(smem_storage + offset_o_tiles_stage0);
    // __hip_bfloat16* o_tiles_sh_stage1 = (__hip_bfloat16*)(smem_storage + offset_o_tiles_stage1); // Fixed here
    // float* acc_o_scales_sh_stage0 = (float*)(smem_storage + offset_acc_o_scales_stage0);
    // float* acc_o_scales_sh_stage1 = (float*)(smem_storage + offset_acc_o_scales_stage1);


    // ===================================================================================
    // INITIAL LOAD: 't' and 'real_token_index' (not pipelined, loaded all at once)
    // ===================================================================================
    const int* t_global_ptr = t_ptr + pid_q_j * stride_tn;
    // Load all 't' and derived 'real_token_index' values for this token into shared memory
    for (int k = 0; k < TOPK; ++k) {
        // Load 't'
        int current_t = t_global_ptr[k * stride_tk];
        t_shared[tid * (TOPK + SMEM_PAD_INT_OR_FLOAT) + k] = current_t;

        // Load 'real_token_index' only if 't' is valid
        if (current_t != -1) {
            real_token_index_shared[tid * (TOPK + SMEM_PAD_INT_OR_FLOAT) + k] = token_index_mapping_ptr[current_t * stride_tim_b + pid_q_j * stride_tim_n];
        } else {
            // Set a safe default if t is -1 to avoid invalid memory access later
            real_token_index_shared[tid * (TOPK + SMEM_PAD_INT_OR_FLOAT) + k] = 0; // Or any sentinel value
        }
    }
    __syncthreads(); // Ensure all 't' and 'real_token_index' are loaded

    // ===================================================================================
    // INITIALIZATION: Load initial 'o' and 'lse' state
    // ===================================================================================

    float lse = lse_ptr[pid_q_j * stride_lse_n];
    const float m_ij_last = m_ij_last_ptr[pid_q_j];

    // ===================================================================================
    // SOFTWARE PIPELINING LOOP
    // ===================================================================================
    int current_stage = 0;
    int next_stage = 1;

    // Prefetch for the first iteration (block_id = 0)
    // This part runs while the main loop would normally be doing computation
    // but there's no prior computation to overlap with.
    {
        int t_pref = t_shared[tid * (TOPK + SMEM_PAD_INT_OR_FLOAT) + 0];
        int real_token_index_pref = real_token_index_shared[tid * (TOPK + SMEM_PAD_INT_OR_FLOAT) + 0];

        // Pointers for current prefetch stage
        float* m_ij_sh_curr_pref = (current_stage == 0) ? m_ij_sh_stage0 : m_ij_sh_stage1;
        float* l_ij_sh_curr_pref = (current_stage == 0) ? l_ij_sh_stage0 : l_ij_sh_stage1;

        if (t_pref != -1) {
            // Determine global pointers based on t_pref and real_token_index_pref
            int real_block_pos_pref;
            const float* l_ij_ptr_pref;
            int64_t stride_l_ij_b_pref, stride_l_ij_n_pref;

            if (t_pref == 0) {
                real_block_pos_pref = 0;
                l_ij_ptr_pref = l_ij_first_ptr;

                stride_l_ij_b_pref = stride_l_ij_fb; stride_l_ij_n_pref = stride_l_ij_fn;

            } else {
                real_block_pos_pref = t_pref - 1;
                l_ij_ptr_pref = l_ij_rest_ptr;

                stride_l_ij_b_pref = stride_l_ij_rb; stride_l_ij_n_pref = stride_l_ij_rn;

            }

            // Load scalars for this thread and store to shared memory
            m_ij_sh_curr_pref[tid] = m_ij_ptr[t_pref * stride_m_ij_b + pid_q_j * stride_m_ij_n];
            l_ij_sh_curr_pref[tid] = l_ij_ptr_pref[real_block_pos_pref * stride_l_ij_b_pref + real_token_index_pref * stride_l_ij_n_pref];


        } else {
            // If t_pref is -1, fill shared memory with safe defaults
            m_ij_sh_curr_pref[tid] = 0.0f;
            l_ij_sh_curr_pref[tid] = 0.0f;
        }
    } // End of initial prefetch block

    // Main pipelined loop
    for (int block_id = 0; block_id < TOPK; ++block_id) {
        __syncthreads(); // Sync after prefetch/before compute

        // Pointers for current compute stage (using data that was prefetched)
        float* m_ij_sh_curr_comp = (current_stage == 0) ? m_ij_sh_stage0 : m_ij_sh_stage1;
        float* l_ij_sh_curr_comp = (current_stage == 0) ? l_ij_sh_stage0 : l_ij_sh_stage1;

        // Get 't' from shared memory for this block_id
        const int t = t_shared[tid * (TOPK + SMEM_PAD_INT_OR_FLOAT) + block_id];

        if (t != -1) {
            // Get data for current compute iteration from shared memory
            const float m_ij = m_ij_sh_curr_comp[tid];
            const float l_ij = l_ij_sh_curr_comp[tid];

            // --- LSE Update ---
            const float delta = lse - m_ij;
            const float log_delta = exp2_fast(delta) + l_ij;
            lse = m_ij + log2_fast(log_delta);

        }

        // Prefetch data for the NEXT iteration (block_id + 1)
        if (block_id + 1 < TOPK) {
            int t_next = t_shared[tid * (TOPK + SMEM_PAD_INT_OR_FLOAT) + block_id + 1];
            int real_token_index_next = real_token_index_shared[tid * (TOPK + SMEM_PAD_INT_OR_FLOAT) + block_id + 1];

            // Pointers for next prefetch stage
            float* m_ij_sh_next_pref = (next_stage == 0) ? m_ij_sh_stage0 : m_ij_sh_stage1;
            float* l_ij_sh_next_pref = (next_stage == 0) ? l_ij_sh_stage0 : l_ij_sh_stage1;

            if (t_next != -1) {
                // Determine global pointers based on t_next and real_token_index_next
                int real_block_pos_next;
                const float* l_ij_ptr_next;
                int64_t stride_l_ij_b_next, stride_l_ij_n_next;

                if (t_next == 0) {
                    real_block_pos_next = 0;
                    l_ij_ptr_next = l_ij_first_ptr;
                    stride_l_ij_b_next = stride_l_ij_fb; stride_l_ij_n_next = stride_l_ij_fn;

                } else {
                    real_block_pos_next = t_next - 1;
                    l_ij_ptr_next = l_ij_rest_ptr;

                    stride_l_ij_b_next = stride_l_ij_rb; stride_l_ij_n_next = stride_l_ij_rn;

                }

                // Load scalars for this thread and store to shared memory
                m_ij_sh_next_pref[tid] = m_ij_ptr[t_next * stride_m_ij_b + pid_q_j * stride_m_ij_n];
                l_ij_sh_next_pref[tid] = l_ij_ptr_next[real_block_pos_next * stride_l_ij_b_next + real_token_index_next * stride_l_ij_n_next];

            } else {
                // If t_next is -1, fill shared memory with safe defaults
                m_ij_sh_next_pref[tid] = 0.0f;
                l_ij_sh_next_pref[tid] = 0.0f;
            }
        }

        // Swap stages for the next iteration
        current_stage = 1 - current_stage;
        next_stage = 1 - next_stage;
    }
    __syncthreads(); // Ensure last iteration's computation is complete

    // ===================================================================================
    // FINALIZATION: Apply final scale and store results
    // ===================================================================================

    lse_ptr[pid_q_j * stride_lse_n] = lse;
}


template<int BLOCK_SIZE_D, int TOPK>
__global__ void o_reduce_kernel_cuda(
    // Pointers
    float* lse_ptr,
    const float* m_ij_last_ptr,
    __hip_bfloat16* o_ptr,
    const __hip_bfloat16* o_tiles_first_ptr,
    const __hip_bfloat16* o_tiles_rest_ptr,
    const float* acc_o_scales_first_ptr,
    const float* acc_o_scales_rest_ptr,
    const int* t_ptr,
    const int* token_index_mapping_ptr,
    // Scalars
    int start_head_id,
    int total_len,
    // Strides
    int64_t stride_lse_n,
    int64_t stride_on, int64_t stride_od,
    int64_t stride_otfb, int64_t stride_otfn, int64_t stride_otfd,
    int64_t stride_otrb, int64_t stride_otrn, int64_t stride_otrd,
    int64_t stride_acc_fb, int64_t stride_acc_fn,
    int64_t stride_acc_rb, int64_t stride_acc_rn,
    int64_t stride_tn, int64_t stride_tk,
    int64_t stride_tim_b, int64_t stride_tim_n
) {
    // ===================================================================================
    // THREAD AND BLOCK CONFIGURATION
    // ===================================================================================
    constexpr int WARP_SIZE = 32;
    constexpr int THREADS_PER_TOKEN = 32;
    constexpr int TOKENS_PER_BLOCK = 4;  // 128 threads / 32 threads per token
    constexpr int VALUES_PER_THREAD = BLOCK_SIZE_D / WARP_SIZE; // 128/32 = 4
    
    const int tid = threadIdx.x;
    const int token_local_id = tid / THREADS_PER_TOKEN; // 0-3, which token within block
    const int thread_in_token = tid % THREADS_PER_TOKEN; // 0-31, thread within token
    
    const int base_token_id = blockIdx.x * TOKENS_PER_BLOCK;
    
    // ===================================================================================
    // SHARED MEMORY SETUP FOR DOUBLE BUFFERING
    // ===================================================================================
    extern __shared__ char smem_storage[];
    
    size_t offset = 0;
    
    // Static data (loaded once per token)
    size_t offset_t_shared = offset;
    offset += sizeof(int) * TOKENS_PER_BLOCK * TOPK;
    
    size_t offset_real_token_index_shared = offset;
    offset += sizeof(int) * TOKENS_PER_BLOCK * TOPK;
    
    // Double buffered data - Stage 0
    size_t offset_acc_o_scales_stage0 = offset;
    offset += sizeof(float) * TOKENS_PER_BLOCK;
    
    size_t offset_o_tiles_stage0 = offset;
    offset += sizeof(float) * TOKENS_PER_BLOCK * BLOCK_SIZE_D; // Use float in shared memory
    
    // Double buffered data - Stage 1
    size_t offset_acc_o_scales_stage1 = offset;
    offset += sizeof(float) * TOKENS_PER_BLOCK;
    
    size_t offset_o_tiles_stage1 = offset;
    offset += sizeof(float) * TOKENS_PER_BLOCK * BLOCK_SIZE_D;
    
    // Cast shared memory pointers
    int* t_shared = (int*)(smem_storage + offset_t_shared);
    int* real_token_index_shared = (int*)(smem_storage + offset_real_token_index_shared);
    
    float* acc_o_scales_sh_stage0 = (float*)(smem_storage + offset_acc_o_scales_stage0);
    float* acc_o_scales_sh_stage1 = (float*)(smem_storage + offset_acc_o_scales_stage1);
    float* o_tiles_sh_stage0 = (float*)(smem_storage + offset_o_tiles_stage0);
    float* o_tiles_sh_stage1 = (float*)(smem_storage + offset_o_tiles_stage1);
    
    // ===================================================================================
    // LOOP OVER TOKENS IN THIS BLOCK
    // ===================================================================================
    for (int token_offset = 0; token_offset < TOKENS_PER_BLOCK; ++token_offset) {
        const int token_id = base_token_id + token_offset;
        
        // Early exit if token_id exceeds total_len
        if (token_id >= total_len) {
            break;
        }
        
        // Only threads assigned to this token participate
        if (token_local_id == token_offset) {
            
            // ===================================================================================
            // STATIC DATA LOADING: Load 't' and 'real_token_index' once
            // ===================================================================================
            const int* t_global_ptr = t_ptr + token_id * stride_tn;
            
            // Thread 0 loads all t values for this token
            if (thread_in_token == 0) {
                for (int k = 0; k < TOPK; ++k) {
                    int current_t = t_global_ptr[k * stride_tk];
                    t_shared[token_offset * TOPK + k] = current_t;
                    
                    if (current_t != -1) {
                        real_token_index_shared[token_offset * TOPK + k] = 
                            token_index_mapping_ptr[current_t * stride_tim_b + token_id * stride_tim_n];
                    } else {
                        real_token_index_shared[token_offset * TOPK + k] = 0;
                    }
                }
            }
            
            // ===================================================================================
            // INITIALIZATION: Load initial 'o' state and constants
            // ===================================================================================
            float acc_o[VALUES_PER_THREAD];
            __hip_bfloat16* o_local_ptr = o_ptr + token_id * stride_on;
            
            // Each thread loads its portion of BLOCK_SIZE_D
            for (int d_idx = 0; d_idx < VALUES_PER_THREAD; ++d_idx) {
                int d = thread_in_token * VALUES_PER_THREAD + d_idx;
                if (d < BLOCK_SIZE_D) {
                    acc_o[d_idx] = __bfloat162float(o_local_ptr[d * stride_od]);
                } else {
                    acc_o[d_idx] = 0.0f;
                }
            }
            
            // Thread 0 loads constants (other threads will read from shared memory later)
            __shared__ float shared_lse[TOKENS_PER_BLOCK];
            __shared__ float shared_m_ij_last[TOKENS_PER_BLOCK];
            __shared__ float shared_final_scale[TOKENS_PER_BLOCK];
            
            if (thread_in_token == 0) {
                float lse = lse_ptr[token_id * stride_lse_n];
                float m_ij_last = m_ij_last_ptr[token_id];
                
                shared_lse[token_offset] = lse;
                shared_m_ij_last[token_offset] = m_ij_last;
                shared_final_scale[token_offset] = exp2f(m_ij_last - lse);
            }
            
            __syncthreads(); // Wait for static data loading
            
            // ===================================================================================
            // SOFTWARE PIPELINING LOOP WITH DOUBLE BUFFERING
            // ===================================================================================
            int current_stage = 0;
            
            // Prefetch for the first iteration (block_id = 0)
            if (TOPK > 0) {
                int t_pref = t_shared[token_offset * TOPK + 0];
                int real_token_index_pref = real_token_index_shared[token_offset * TOPK + 0];
                
                // Choose stage 0 for initial prefetch
                float* acc_o_scales_sh_curr = (current_stage == 0) ? acc_o_scales_sh_stage0 : acc_o_scales_sh_stage1;
                float* o_tiles_sh_curr = (current_stage == 0) ? o_tiles_sh_stage0 : o_tiles_sh_stage1;
                
                if (t_pref != -1) {
                    // Determine pointers and strides
                    int real_block_pos_pref;
                    const __hip_bfloat16* o_tiles_ptr_pref;
                    const float* acc_o_scales_ptr_pref;
                    int64_t stride_otb_pref, stride_otn_pref, stride_otd_pref;
                    int64_t stride_acc_b_pref, stride_acc_n_pref;
                    
                    if (t_pref == 0) {
                        real_block_pos_pref = 0;
                        o_tiles_ptr_pref = o_tiles_first_ptr;
                        acc_o_scales_ptr_pref = acc_o_scales_first_ptr;
                        stride_otb_pref = stride_otfb; stride_otn_pref = stride_otfn; stride_otd_pref = stride_otfd;
                        stride_acc_b_pref = stride_acc_fb; stride_acc_n_pref = stride_acc_fn;
                    } else {
                        real_block_pos_pref = t_pref - 1;
                        o_tiles_ptr_pref = o_tiles_rest_ptr;
                        acc_o_scales_ptr_pref = acc_o_scales_rest_ptr;
                        stride_otb_pref = stride_otrb; stride_otn_pref = stride_otrn; stride_otd_pref = stride_otrd;
                        stride_acc_b_pref = stride_acc_rb; stride_acc_n_pref = stride_acc_rn;
                    }
                    
                    // Thread 0 loads scalar
                    if (thread_in_token == 0) {
                        acc_o_scales_sh_curr[token_offset] = acc_o_scales_ptr_pref[real_block_pos_pref * stride_acc_b_pref + real_token_index_pref * stride_acc_n_pref];
                    }
                    
                    // All threads cooperatively load o_tiles
                    const __hip_bfloat16* o_tiles_local_ptr_pref = o_tiles_ptr_pref + 
                        real_block_pos_pref * stride_otb_pref + real_token_index_pref * stride_otn_pref;
                    
                    for (int d_idx = 0; d_idx < VALUES_PER_THREAD; ++d_idx) {
                        int d = thread_in_token * VALUES_PER_THREAD + d_idx;
                        if (d < BLOCK_SIZE_D) {
                            // Convert bf16 to float when loading into shared memory
                            o_tiles_sh_curr[token_offset * BLOCK_SIZE_D + d] = 
                                __bfloat162float(o_tiles_local_ptr_pref[d * stride_otd_pref]);
                        }
                    }
                } else {
                    // If t_pref is -1, fill with safe defaults
                    if (thread_in_token == 0) {
                        acc_o_scales_sh_curr[token_offset] = 0.0f;
                    }
                    
                    for (int d_idx = 0; d_idx < VALUES_PER_THREAD; ++d_idx) {
                        int d = thread_in_token * VALUES_PER_THREAD + d_idx;
                        if (d < BLOCK_SIZE_D) {
                            o_tiles_sh_curr[token_offset * BLOCK_SIZE_D + d] = 0.0f;
                        }
                    }
                }
            }
            
            // Main pipelined loop
            for (int block_id = 0; block_id < TOPK; ++block_id) {
                __syncthreads(); // Sync after prefetch/before compute
                
                // Choose current compute stage
                float* acc_o_scales_sh_comp = (current_stage == 0) ? acc_o_scales_sh_stage0 : acc_o_scales_sh_stage1;
                float* o_tiles_sh_comp = (current_stage == 0) ? o_tiles_sh_stage0 : o_tiles_sh_stage1;
                
                // Get 't' from shared memory for this block_id
                const int t = t_shared[token_offset * TOPK + block_id];
                
                if (t != -1) {
                    // ===================================================================================
                    // COMPUTE: Update accumulator with current tile
                    // ===================================================================================
                    const float acc_o_scale_tile = acc_o_scales_sh_comp[token_offset];
                    
                    for (int d_idx = 0; d_idx < VALUES_PER_THREAD; ++d_idx) {
                        int d = thread_in_token * VALUES_PER_THREAD + d_idx;
                        if (d < BLOCK_SIZE_D) {
                            float o_tile_val = o_tiles_sh_comp[token_offset * BLOCK_SIZE_D + d];
                            // Update accumulator: o_tile + acc_o * scale
                            acc_o[d_idx] = o_tile_val + acc_o[d_idx] * acc_o_scale_tile;
                        }
                    }
                }
                
                // ===================================================================================
                // PREFETCH: Load data for next iteration (if exists)
                // ===================================================================================
                if (block_id + 1 < TOPK) {
                    int next_stage = 1 - current_stage;
                    int t_next = t_shared[token_offset * TOPK + (block_id + 1)];
                    int real_token_index_next = real_token_index_shared[token_offset * TOPK + (block_id + 1)];
                    
                    // Choose next stage for prefetch
                    float* acc_o_scales_sh_next = (next_stage == 0) ? acc_o_scales_sh_stage0 : acc_o_scales_sh_stage1;
                    float* o_tiles_sh_next = (next_stage == 0) ? o_tiles_sh_stage0 : o_tiles_sh_stage1;
                    
                    if (t_next != -1) {
                        // Determine pointers and strides for next iteration
                        int real_block_pos_next;
                        const __hip_bfloat16* o_tiles_ptr_next;
                        const float* acc_o_scales_ptr_next;
                        int64_t stride_otb_next, stride_otn_next, stride_otd_next;
                        int64_t stride_acc_b_next, stride_acc_n_next;
                        
                        if (t_next == 0) {
                            real_block_pos_next = 0;
                            o_tiles_ptr_next = o_tiles_first_ptr;
                            acc_o_scales_ptr_next = acc_o_scales_first_ptr;
                            stride_otb_next = stride_otfb; stride_otn_next = stride_otfn; stride_otd_next = stride_otfd;
                            stride_acc_b_next = stride_acc_fb; stride_acc_n_next = stride_acc_fn;
                        } else {
                            real_block_pos_next = t_next - 1;
                            o_tiles_ptr_next = o_tiles_rest_ptr;
                            acc_o_scales_ptr_next = acc_o_scales_rest_ptr;
                            stride_otb_next = stride_otrb; stride_otn_next = stride_otrn; stride_otd_next = stride_otrd;
                            stride_acc_b_next = stride_acc_rb; stride_acc_n_next = stride_acc_rn;
                        }
                        
                        // Thread 0 loads scalar for next iteration
                        if (thread_in_token == 0) {
                            acc_o_scales_sh_next[token_offset] = acc_o_scales_ptr_next[real_block_pos_next * stride_acc_b_next + real_token_index_next * stride_acc_n_next];
                        }
                        
                        // All threads cooperatively load o_tiles for next iteration
                        const __hip_bfloat16* o_tiles_local_ptr_next = o_tiles_ptr_next + 
                            real_block_pos_next * stride_otb_next + real_token_index_next * stride_otn_next;
                        
                        for (int d_idx = 0; d_idx < VALUES_PER_THREAD; ++d_idx) {
                            int d = thread_in_token * VALUES_PER_THREAD + d_idx;
                            if (d < BLOCK_SIZE_D) {
                                // Convert bf16 to float when loading into shared memory
                                o_tiles_sh_next[token_offset * BLOCK_SIZE_D + d] = 
                                    __bfloat162float(o_tiles_local_ptr_next[d * stride_otd_next]);
                            }
                        }
                    } else {
                        // If t_next is -1, fill with safe defaults
                        if (thread_in_token == 0) {
                            acc_o_scales_sh_next[token_offset] = 0.0f;
                        }
                        
                        for (int d_idx = 0; d_idx < VALUES_PER_THREAD; ++d_idx) {
                            int d = thread_in_token * VALUES_PER_THREAD + d_idx;
                            if (d < BLOCK_SIZE_D) {
                                o_tiles_sh_next[token_offset * BLOCK_SIZE_D + d] = 0.0f;
                            }
                        }
                    }
                }
                
                // Flip stage for next iteration
                current_stage = 1 - current_stage;
            } // End of main pipelined loop
            
            // ===================================================================================
            // FINALIZATION: Apply final scale and write back results
            // ===================================================================================
            const float final_scale = shared_final_scale[token_offset];
            
            // Apply final scaling and write back to global memory
            for (int d_idx = 0; d_idx < VALUES_PER_THREAD; ++d_idx) {
                int d = thread_in_token * VALUES_PER_THREAD + d_idx;
                if (d < BLOCK_SIZE_D) {
                    acc_o[d_idx] *= final_scale;
                    o_local_ptr[d * stride_od] = __float2bfloat16_rn(acc_o[d_idx]);
                }
            }
            
        } // End of token_local_id == token_offset check
        
        __syncthreads(); // Sync between different tokens in the block
        
    } // End of token loop
}

// // Naive
// template<int BLOCK_SIZE_D, int TOPK>
// __global__ void o_reduce_kernel_cuda(
//     // Pointers
//     float* lse_ptr,
//     const float* m_ij_last_ptr,
//     __hip_bfloat16* o_ptr,                  // <-- CORRECTED TYPE
//     const __hip_bfloat16* o_tiles_first_ptr,  // <-- CORRECTED TYPE
//     const __hip_bfloat16* o_tiles_rest_ptr,   // <-- CORRECTED TYPE
//     const float* acc_o_scales_first_ptr,
//     const float* acc_o_scales_rest_ptr,
//     const int* t_ptr,
//     const int* token_index_mapping_ptr,
//     // Scalars
//     int start_head_id,
//     int total_len,
//     // Strides
//     int64_t stride_lse_n,
//     int64_t stride_on, int64_t stride_od,
//     int64_t stride_otfb, int64_t stride_otfn, int64_t stride_otfd,
//     int64_t stride_otrb, int64_t stride_otrn, int64_t stride_otrd,
//     int64_t stride_acc_fb, int64_t stride_acc_fn,
//     int64_t stride_acc_rb, int64_t stride_acc_rn,
//     int64_t stride_tn, int64_t stride_tk,
//     int64_t stride_tim_b, int64_t stride_tim_n
// ) {
//     // ===================================================================================
//     // KERNEL SETUP: Map threads to tokens
//     // ===================================================================================
//     const int tid = threadIdx.x;
//     const int pid_q_j = blockIdx.x * blockDim.x + tid;

//     if (pid_q_j >= total_len) {
//         return;
//     }

//     // ===================================================================================
//     // OPTIMIZATION: Bulk load 't' indices into shared memory
//     // ===================================================================================
//     extern __shared__ int t_shared_storage[]; // Dynamically sized shared memory
//     int* t_shared = t_shared_storage; // [BLOCK_DIM_X][TOPK]

//     const int* t_global_ptr = t_ptr + pid_q_j * stride_tn;

//     // Each thread loads its own TOPK indices into its row in shared memory.
//     // This can be further optimized for coalescing, but is a correct starting point.
//     for (int k = 0; k < TOPK; ++k) {
//         t_shared[tid * TOPK + k] = t_global_ptr[k * stride_tk];
        
//     }
//     __syncthreads(); // Ensure all 't' indices are loaded before proceeding.

//     // ===================================================================================
//     // INITIALIZATION: Load initial 'o' and 'lse' state
//     // ===================================================================================
//     // Accumulator for 'o' must be in high precision (float32)
//     float acc_o[BLOCK_SIZE_D];

//     // Load initial 'o' values, converting from bf16 to float32
//     __hip_bfloat16* o_local_ptr = o_ptr + pid_q_j * stride_on;
//     for (int d = 0; d < BLOCK_SIZE_D; ++d) {
//         if (d * stride_od < total_len * stride_on) { // Boundary check
//             acc_o[d] = __bfloat162float(o_local_ptr[d * stride_od]);
//         } else {
//             acc_o[d] = 0.0f;
//         }
//     }

//     float lse = lse_ptr[pid_q_j * stride_lse_n];
//     const float m_ij_last = m_ij_last_ptr[pid_q_j];

//     // ===================================================================================
//     // MAIN REDUCTION LOOP
//     // ===================================================================================
//     for (int block_id = 0; block_id < TOPK; ++block_id) {
//         const int t = t_shared[tid * TOPK + block_id];

//         if (t != -1) {
//             // Triton's branching logic translated to CUDA
//             int real_block_pos;
            
//             const __hip_bfloat16* o_tiles_ptr;
//             const float* acc_o_scales_ptr;
            
//             int64_t stride_otb, stride_otn;
//             int64_t stride_acc_b, stride_acc_n;

//             if (t == 0) {
//                 real_block_pos = 0;
                
//                 o_tiles_ptr = o_tiles_first_ptr;
//                 acc_o_scales_ptr = acc_o_scales_first_ptr;
                
//                 stride_otb = stride_otfb; stride_otn = stride_otfn;
//                 stride_acc_b = stride_acc_fb; stride_acc_n = stride_acc_fn;
//             } else {
//                 real_block_pos = t - 1;
                
//                 o_tiles_ptr = o_tiles_rest_ptr;
//                 acc_o_scales_ptr = acc_o_scales_rest_ptr;
                
//                 stride_otb = stride_otrb; stride_otn = stride_otrn;
//                 stride_acc_b = stride_acc_rb; stride_acc_n = stride_acc_rn;
//             }

//             const int real_token_index = token_index_mapping_ptr[t * stride_tim_b + pid_q_j * stride_tim_n];

//             // --- Vector loads and 'o' update ---
//             const __hip_bfloat16* o_tiles_local_ptr = o_tiles_ptr + real_block_pos * stride_otb + real_token_index * stride_otn;
//             const float acc_o_scale_tile = acc_o_scales_ptr[real_block_pos * stride_acc_b + real_token_index * stride_acc_n];

//             // This loop loads o_tiles (bf16), converts to float, and updates acc_o (float)
//             for (int d = 0; d < BLOCK_SIZE_D; ++d) {
//                 float o_tile_val = __bfloat162float(o_tiles_local_ptr[d * stride_otfd]);
//                 acc_o[d] = o_tile_val + acc_o[d] * acc_o_scale_tile;
//             }
//         }
//     }

//     // ===================================================================================
//     // FINALIZATION: Apply final scale and store results
//     // ===================================================================================
//     const float final_scale = exp2_fast(m_ij_last - lse);

//     // Store final 'o', converting from float32 accumulator to bf16 storage
//     for (int d = 0; d < BLOCK_SIZE_D; ++d) {
//          if (d * stride_od < total_len * stride_on) { // Boundary check
//             acc_o[d] *= final_scale;
//             o_local_ptr[d * stride_od] = __float2bfloat16_rn(acc_o[d]);
//         }
//     }

// }



// // Define constants for the kernel.
// constexpr int BLOCK_DIM_X = 128;

// // __device__ functions for math, equivalent to tl.exp2 and tl.log2
// __device__ inline float exp2_fast(float x) {
//     return exp2f(x);
// }

// __device__ inline float log2_fast(float x) {
//     return log2f(x);
// }

// #define WARP_SIZE 32

// // Helper to determine padding for shared memory to avoid bank conflicts
// // For int/float, typically 32 banks. If size is a multiple of 32, add 1.
// #define SMEM_PAD_INT_OR_FLOAT ((WARP_SIZE % 32 == 0) ? 1 : 0)

// template<int BLOCK_SIZE_D, int TOPK>
// __global__ void reduce_kernel_cuda(
//     // Pointers
//     float* lse_ptr,
//     const float* m_ij_ptr,
//     const float* l_ij_first_ptr,
//     const float* l_ij_rest_ptr,
//     const float* m_ij_last_ptr,
//     __hip_bfloat16* o_ptr,
//     const __hip_bfloat16* o_tiles_first_ptr,
//     const __hip_bfloat16* o_tiles_rest_ptr,
//     const float* acc_o_scales_first_ptr,
//     const float* acc_o_scales_rest_ptr,
//     const int* t_ptr,
//     const int* token_index_mapping_ptr,
//     // Scalars
//     int start_head_id,
//     int total_len,
//     // Strides
//     int64_t stride_lse_n,
//     int64_t stride_m_ij_b, int64_t stride_m_ij_n,
//     int64_t stride_l_ij_fb, int64_t stride_l_ij_fn,
//     int64_t stride_l_ij_rb, int64_t stride_l_ij_rn,
//     int64_t stride_on, int64_t stride_od,
//     int64_t stride_otfb, int64_t stride_otfn, int64_t stride_otfd,
//     int64_t stride_otrb, int64_t stride_otrn, int64_t stride_otrd,
//     int64_t stride_acc_fb, int64_t stride_acc_fn,
//     int64_t stride_acc_rb, int64_t stride_acc_rn,
//     int64_t stride_tn, int64_t stride_tk,
//     int64_t stride_tim_b, int64_t stride_tim_n
// ) {
//     // const int tid = threadIdx.x; // 0 to WARP_SIZE-1
//     // const int pid_q_j = blockIdx.x; // Each block handles one token
//     const int tid = threadIdx.x;
//     const int pid_q_j = blockIdx.x * blockDim.x + tid;

//     if (pid_q_j >= total_len) {
//         return;
//     }

//     // ===================================================================================
//     // SHARED MEMORY SETUP
//     // ===================================================================================
//     extern __shared__ char smem_storage[]; // Dynamically sized shared memory

//     // Calculate offsets for different shared memory regions
//     // All sizes are per-thread * padded.
//     size_t offset_t_shared = 0;
//     size_t offset_real_token_index_shared = offset_t_shared + sizeof(int) * WARP_SIZE * (TOPK + SMEM_PAD_INT_OR_FLOAT);

//     // Two stages for pipelined data
//     size_t offset_m_ij_stage0 = offset_real_token_index_shared + sizeof(int) * WARP_SIZE * (TOPK + SMEM_PAD_INT_OR_FLOAT);
//     size_t offset_m_ij_stage1 = offset_m_ij_stage0 + sizeof(float) * WARP_SIZE; // One float per thread per stage

//     size_t offset_l_ij_stage0 = offset_m_ij_stage1 + sizeof(float) * WARP_SIZE;
//     size_t offset_l_ij_stage1 = offset_l_ij_stage0 + sizeof(float) * WARP_SIZE;

//     // BLOCK_SIZE_D elements for o_tiles. Each thread loads BLOCK_SIZE_D / WARP_SIZE.
//     // Total size is BLOCK_SIZE_D * sizeof(__hip_bfloat16) * 2 stages
//     // We need to ensure o_tiles_sh_stage0/1 are aligned for bf16 access.
//     // Assuming BLOCK_SIZE_D is a multiple of WARP_SIZE for simplicity.
//     // Padding here is for the entire BLOCK_SIZE_D if needed, not per-thread
//     size_t offset_o_tiles_stage0 = offset_l_ij_stage1 + sizeof(float) * WARP_SIZE;
//     size_t offset_o_tiles_stage1 = offset_o_tiles_stage0 + sizeof(__hip_bfloat16) * BLOCK_SIZE_D;

//     size_t offset_acc_o_scales_stage0 = offset_o_tiles_stage1 + sizeof(__hip_bfloat16) * BLOCK_SIZE_D;
//     size_t offset_acc_o_scales_stage1 = offset_acc_o_scales_stage0 + sizeof(float) * WARP_SIZE;

//     // Pointers to shared memory regions
//     int* t_shared = (int*)(smem_storage + offset_t_shared);
//     int* real_token_index_shared = (int*)(smem_storage + offset_real_token_index_shared);

//     float* m_ij_sh_stage0 = (float*)(smem_storage + offset_m_ij_stage0);
//     float* m_ij_sh_stage1 = (float*)(smem_storage + offset_m_ij_stage1);
//     float* l_ij_sh_stage0 = (float*)(smem_storage + offset_l_ij_stage0);
//     float* l_ij_sh_stage1 = (float*)(smem_storage + offset_l_ij_stage1);
//     __hip_bfloat16* o_tiles_sh_stage0 = (__hip_bfloat16*)(smem_storage + offset_o_tiles_stage0);
//     __hip_bfloat16* o_tiles_sh_stage1 = (__hip_bfloat16*)(smem_storage + offset_o_tiles_stage1); // Fixed here
//     float* acc_o_scales_sh_stage0 = (float*)(smem_storage + offset_acc_o_scales_stage0);
//     float* acc_o_scales_sh_stage1 = (float*)(smem_storage + offset_acc_o_scales_stage1);


//     // ===================================================================================
//     // INITIAL LOAD: 't' and 'real_token_index' (not pipelined, loaded all at once)
//     // ===================================================================================
//     const int* t_global_ptr = t_ptr + pid_q_j * stride_tn;
//     // Load all 't' and derived 'real_token_index' values for this token into shared memory
//     for (int k = 0; k < TOPK; ++k) {
//         // Load 't'
//         int current_t = t_global_ptr[k * stride_tk];
//         t_shared[tid * (TOPK + SMEM_PAD_INT_OR_FLOAT) + k] = current_t;

//         // Load 'real_token_index' only if 't' is valid
//         if (current_t != -1) {
//             real_token_index_shared[tid * (TOPK + SMEM_PAD_INT_OR_FLOAT) + k] = token_index_mapping_ptr[current_t * stride_tim_b + pid_q_j * stride_tim_n];
//         } else {
//             // Set a safe default if t is -1 to avoid invalid memory access later
//             real_token_index_shared[tid * (TOPK + SMEM_PAD_INT_OR_FLOAT) + k] = 0; // Or any sentinel value
//         }
//     }
//     __syncthreads(); // Ensure all 't' and 'real_token_index' are loaded

//     // ===================================================================================
//     // INITIALIZATION: Load initial 'o' and 'lse' state
//     // ===================================================================================
//     // Each thread handles a portion of BLOCK_SIZE_D
//     constexpr int TILE_SIZE_D = BLOCK_SIZE_D; // Assumes BLOCK_SIZE_D is multiple of WARP_SIZE
//     float acc_o[TILE_SIZE_D];

//     __hip_bfloat16* o_local_ptr = o_ptr + pid_q_j * stride_on;
//     for (int d_idx = 0; d_idx < TILE_SIZE_D; ++d_idx) {
//         // int d = tid * TILE_SIZE_D + d_idx; // Global 'd' index
//         if (d_idx < BLOCK_SIZE_D) {
//             acc_o[d_idx] = __bfloat162float(o_local_ptr[d_idx * stride_od]);
//         } else {
//             acc_o[d_idx] = 0.0f; // Should not happen if BLOCK_SIZE_D is multiple of WARP_SIZE
//         }
//     }

//     float lse = lse_ptr[pid_q_j * stride_lse_n];
//     const float m_ij_last = m_ij_last_ptr[pid_q_j];

//     // ===================================================================================
//     // SOFTWARE PIPELINING LOOP
//     // ===================================================================================
//     int current_stage = 0;
//     int next_stage = 1;

//     // Prefetch for the first iteration (block_id = 0)
//     // This part runs while the main loop would normally be doing computation
//     // but there's no prior computation to overlap with.
//     {
//         int t_pref = t_shared[tid * (TOPK + SMEM_PAD_INT_OR_FLOAT) + 0];
//         int real_token_index_pref = real_token_index_shared[tid * (TOPK + SMEM_PAD_INT_OR_FLOAT) + 0];

//         // Pointers for current prefetch stage
//         float* m_ij_sh_curr_pref = (current_stage == 0) ? m_ij_sh_stage0 : m_ij_sh_stage1;
//         float* l_ij_sh_curr_pref = (current_stage == 0) ? l_ij_sh_stage0 : l_ij_sh_stage1;
//         __hip_bfloat16* o_tiles_sh_curr_pref = (current_stage == 0) ? o_tiles_sh_stage0 : o_tiles_sh_stage1;
//         float* acc_o_scales_sh_curr_pref = (current_stage == 0) ? acc_o_scales_sh_stage0 : acc_o_scales_sh_stage1;

//         if (t_pref != -1) {
//             // Determine global pointers based on t_pref and real_token_index_pref
//             int real_block_pos_pref;
//             const float* l_ij_ptr_pref;
//             const __hip_bfloat16* o_tiles_ptr_pref;
//             const float* acc_o_scales_ptr_pref;
//             int64_t stride_l_ij_b_pref, stride_l_ij_n_pref;
//             int64_t stride_otb_pref, stride_otn_pref;
//             int64_t stride_acc_b_pref, stride_acc_n_pref;

//             if (t_pref == 0) {
//                 real_block_pos_pref = 0;
//                 l_ij_ptr_pref = l_ij_first_ptr;
//                 o_tiles_ptr_pref = o_tiles_first_ptr;
//                 acc_o_scales_ptr_pref = acc_o_scales_first_ptr;
//                 stride_l_ij_b_pref = stride_l_ij_fb; stride_l_ij_n_pref = stride_l_ij_fn;
//                 stride_otb_pref = stride_otfb; stride_otn_pref = stride_otfn;
//                 stride_acc_b_pref = stride_acc_fb; stride_acc_n_pref = stride_acc_fn;
//             } else {
//                 real_block_pos_pref = t_pref - 1;
//                 l_ij_ptr_pref = l_ij_rest_ptr;
//                 o_tiles_ptr_pref = o_tiles_rest_ptr;
//                 acc_o_scales_ptr_pref = acc_o_scales_rest_ptr;
//                 stride_l_ij_b_pref = stride_l_ij_rb; stride_l_ij_n_pref = stride_l_ij_rn;
//                 stride_otb_pref = stride_otrb; stride_otn_pref = stride_otrn;
//                 stride_acc_b_pref = stride_acc_rb; stride_acc_n_pref = stride_acc_rn;
//             }

//             // Load scalars for this thread and store to shared memory
//             m_ij_sh_curr_pref[tid] = m_ij_ptr[t_pref * stride_m_ij_b + pid_q_j * stride_m_ij_n];
//             l_ij_sh_curr_pref[tid] = l_ij_ptr_pref[real_block_pos_pref * stride_l_ij_b_pref + real_token_index_pref * stride_l_ij_n_pref];

//             // Cooperative load for o_tiles and acc_o_scales
//             const __hip_bfloat16* o_tiles_local_ptr_pref = o_tiles_ptr_pref + real_block_pos_pref * stride_otb_pref + real_token_index_pref * stride_otn_pref;
//             const float acc_o_scale_tile_pref = acc_o_scales_ptr_pref[real_block_pos_pref * stride_acc_b_pref + real_token_index_pref * stride_acc_n_pref];

//             for (int d_idx = 0; d_idx < TILE_SIZE_D; ++d_idx) {
//                 // int d = tid * TILE_SIZE_D + d_idx;
//                 o_tiles_sh_curr_pref[d_idx] = o_tiles_local_ptr_pref[d_idx * stride_otfd];
//             }
//             acc_o_scales_sh_curr_pref[tid] = acc_o_scale_tile_pref;
//         } else {
//             // If t_pref is -1, fill shared memory with safe defaults
//             m_ij_sh_curr_pref[tid] = 0.0f;
//             l_ij_sh_curr_pref[tid] = 0.0f;
//             acc_o_scales_sh_curr_pref[tid] = 0.0f;
//             for (int d_idx = 0; d_idx < TILE_SIZE_D; ++d_idx) {
//                 // int d = tid * TILE_SIZE_D + d_idx;
//                 o_tiles_sh_curr_pref[d_idx] = __float2bfloat16(0.0f);
//             }
//         }
//     } // End of initial prefetch block

//     // Main pipelined loop
//     for (int block_id = 0; block_id < TOPK; ++block_id) {
//         __syncthreads(); // Sync after prefetch/before compute

//         // Pointers for current compute stage (using data that was prefetched)
//         float* m_ij_sh_curr_comp = (current_stage == 0) ? m_ij_sh_stage0 : m_ij_sh_stage1;
//         float* l_ij_sh_curr_comp = (current_stage == 0) ? l_ij_sh_stage0 : l_ij_sh_stage1;
//         __hip_bfloat16* o_tiles_sh_curr_comp = (current_stage == 0) ? o_tiles_sh_stage0 : o_tiles_sh_stage1;
//         float* acc_o_scales_sh_curr_comp = (current_stage == 0) ? acc_o_scales_sh_stage0 : acc_o_scales_sh_stage1;

//         // Get 't' from shared memory for this block_id
//         const int t = t_shared[tid * (TOPK + SMEM_PAD_INT_OR_FLOAT) + block_id];

//         if (t != -1) {
//             // Get data for current compute iteration from shared memory
//             const float m_ij = m_ij_sh_curr_comp[tid];
//             const float l_ij = l_ij_sh_curr_comp[tid];
//             const float acc_o_scale_tile = acc_o_scales_sh_curr_comp[tid];

//             // --- LSE Update ---
//             const float delta = lse - m_ij;
//             const float log_delta = exp2_fast(delta) + l_ij;
//             lse = m_ij + log2_fast(log_delta);

//             // --- 'o' update from shared memory ---
//             for (int d_idx = 0; d_idx < TILE_SIZE_D; ++d_idx) {
//                 // int d = tid * TILE_SIZE_D + d_idx;
//                 float o_tile_val = __bfloat162float(o_tiles_sh_curr_comp[d_idx]);
//                 acc_o[d_idx] = o_tile_val + acc_o[d_idx] * acc_o_scale_tile;
//             }
//         }

//         // Prefetch data for the NEXT iteration (block_id + 1)
//         if (block_id + 1 < TOPK) {
//             int t_next = t_shared[tid * (TOPK + SMEM_PAD_INT_OR_FLOAT) + block_id + 1];
//             int real_token_index_next = real_token_index_shared[tid * (TOPK + SMEM_PAD_INT_OR_FLOAT) + block_id + 1];

//             // Pointers for next prefetch stage
//             float* m_ij_sh_next_pref = (next_stage == 0) ? m_ij_sh_stage0 : m_ij_sh_stage1;
//             float* l_ij_sh_next_pref = (next_stage == 0) ? l_ij_sh_stage0 : l_ij_sh_stage1;
//             __hip_bfloat16* o_tiles_sh_next_pref = (next_stage == 0) ? o_tiles_sh_stage0 : o_tiles_sh_stage1;
//             float* acc_o_scales_sh_next_pref = (next_stage == 0) ? acc_o_scales_sh_stage0 : acc_o_scales_sh_stage1;

//             if (t_next != -1) {
//                 // Determine global pointers based on t_next and real_token_index_next
//                 int real_block_pos_next;
//                 const float* l_ij_ptr_next;
//                 const __hip_bfloat16* o_tiles_ptr_next;
//                 const float* acc_o_scales_ptr_next;
//                 int64_t stride_l_ij_b_next, stride_l_ij_n_next;
//                 int64_t stride_otb_next, stride_otn_next;
//                 int64_t stride_acc_b_next, stride_acc_n_next;

//                 if (t_next == 0) {
//                     real_block_pos_next = 0;
//                     l_ij_ptr_next = l_ij_first_ptr;
//                     o_tiles_ptr_next = o_tiles_first_ptr;
//                     acc_o_scales_ptr_next = acc_o_scales_first_ptr;
//                     stride_l_ij_b_next = stride_l_ij_fb; stride_l_ij_n_next = stride_l_ij_fn;
//                     stride_otb_next = stride_otfb; stride_otn_next = stride_otfn;
//                     stride_acc_b_next = stride_acc_fb; stride_acc_n_next = stride_acc_fn;
//                 } else {
//                     real_block_pos_next = t_next - 1;
//                     l_ij_ptr_next = l_ij_rest_ptr;
//                     o_tiles_ptr_next = o_tiles_rest_ptr;
//                     acc_o_scales_ptr_next = acc_o_scales_rest_ptr;
//                     stride_l_ij_b_next = stride_l_ij_rb; stride_l_ij_n_next = stride_l_ij_rn;
//                     stride_otb_next = stride_otrb; stride_otn_next = stride_otrn;
//                     stride_acc_b_next = stride_acc_rb; stride_acc_n_next = stride_acc_rn;
//                 }

//                 // Load scalars for this thread and store to shared memory
//                 m_ij_sh_next_pref[tid] = m_ij_ptr[t_next * stride_m_ij_b + pid_q_j * stride_m_ij_n];
//                 l_ij_sh_next_pref[tid] = l_ij_ptr_next[real_block_pos_next * stride_l_ij_b_next + real_token_index_next * stride_l_ij_n_next];

//                 // Cooperative load for o_tiles and acc_o_scales
//                 const __hip_bfloat16* o_tiles_local_ptr_next = o_tiles_ptr_next + real_block_pos_next * stride_otb_next + real_token_index_next * stride_otn_next;
//                 const float acc_o_scale_tile_next = acc_o_scales_ptr_next[real_block_pos_next * stride_acc_b_next + real_token_index_next * stride_acc_n_next];

//                 for (int d_idx = 0; d_idx < TILE_SIZE_D; ++d_idx) {
//                     // int d = tid * TILE_SIZE_D + d_idx;
//                     o_tiles_sh_next_pref[d_idx] = o_tiles_local_ptr_next[d_idx * stride_otfd];
//                 }
//                 acc_o_scales_sh_next_pref[tid] = acc_o_scale_tile_next;
//             } else {
//                 // If t_next is -1, fill shared memory with safe defaults
//                 m_ij_sh_next_pref[tid] = 0.0f;
//                 l_ij_sh_next_pref[tid] = 0.0f;
//                 acc_o_scales_sh_next_pref[tid] = 0.0f;
//                 for (int d_idx = 0; d_idx < TILE_SIZE_D; ++d_idx) {
//                     // int d = tid * TILE_SIZE_D + d_idx;
//                     o_tiles_sh_next_pref[d_idx] = __float2bfloat16(0.0f);
//                 }
//             }
//         }

//         // Swap stages for the next iteration
//         current_stage = 1 - current_stage;
//         next_stage = 1 - next_stage;
//     }
//     __syncthreads(); // Ensure last iteration's computation is complete

//     // ===================================================================================
//     // FINALIZATION: Apply final scale and store results
//     // ===================================================================================
//     const float final_scale = exp2_fast(m_ij_last - lse);

//     for (int d_idx = 0; d_idx < TILE_SIZE_D; ++d_idx) {
//         //  int d = tid * TILE_SIZE_D + d_idx;
//          if (d_idx < BLOCK_SIZE_D) { // Check only needed if BLOCK_SIZE_D not multiple of WARP_SIZE

//             acc_o[d_idx] *= final_scale;
//             o_ptr[pid_q_j * stride_on + d_idx * stride_od] = __float2bfloat16_rn(acc_o[d_idx]);
//         }
//     }

//     lse_ptr[pid_q_j * stride_lse_n] = lse;
// }


// Naive correct
// template<int BLOCK_SIZE_D, int TOPK>
// __global__ void reduce_kernel_cuda(
//     // Pointers
//     float* lse_ptr,
//     const float* m_ij_ptr,
//     const float* l_ij_first_ptr,
//     const float* l_ij_rest_ptr,
//     const float* m_ij_last_ptr,
//     __hip_bfloat16* o_ptr,                  // <-- CORRECTED TYPE
//     const __hip_bfloat16* o_tiles_first_ptr,  // <-- CORRECTED TYPE
//     const __hip_bfloat16* o_tiles_rest_ptr,   // <-- CORRECTED TYPE
//     const float* acc_o_scales_first_ptr,
//     const float* acc_o_scales_rest_ptr,
//     const int* t_ptr,
//     const int* token_index_mapping_ptr,
//     // Scalars
//     int start_head_id,
//     int total_len,
//     // Strides
//     int64_t stride_lse_n,
//     int64_t stride_m_ij_b, int64_t stride_m_ij_n,
//     int64_t stride_l_ij_fb, int64_t stride_l_ij_fn,
//     int64_t stride_l_ij_rb, int64_t stride_l_ij_rn,
//     int64_t stride_on, int64_t stride_od,
//     int64_t stride_otfb, int64_t stride_otfn, int64_t stride_otfd,
//     int64_t stride_otrb, int64_t stride_otrn, int64_t stride_otrd,
//     int64_t stride_acc_fb, int64_t stride_acc_fn,
//     int64_t stride_acc_rb, int64_t stride_acc_rn,
//     int64_t stride_tn, int64_t stride_tk,
//     int64_t stride_tim_b, int64_t stride_tim_n
// ) {
//     // ===================================================================================
//     // KERNEL SETUP: Map threads to tokens
//     // ===================================================================================
//     const int tid = threadIdx.x;
//     const int pid_q_j = blockIdx.x * blockDim.x + tid;

//     if (pid_q_j >= total_len) {
//         return;
//     }

//     // ===================================================================================
//     // OPTIMIZATION: Bulk load 't' indices into shared memory
//     // ===================================================================================
//     extern __shared__ int t_shared_storage[]; // Dynamically sized shared memory
//     int* t_shared = t_shared_storage; // [BLOCK_DIM_X][TOPK]

//     const int* t_global_ptr = t_ptr + pid_q_j * stride_tn;

//     // Each thread loads its own TOPK indices into its row in shared memory.
//     // This can be further optimized for coalescing, but is a correct starting point.
//     for (int k = 0; k < TOPK; ++k) {
//         t_shared[tid * TOPK + k] = t_global_ptr[k * stride_tk];
        
//     }
//     __syncthreads(); // Ensure all 't' indices are loaded before proceeding.

//     // ===================================================================================
//     // INITIALIZATION: Load initial 'o' and 'lse' state
//     // ===================================================================================
//     // Accumulator for 'o' must be in high precision (float32)
//     float acc_o[BLOCK_SIZE_D];

//     // Load initial 'o' values, converting from bf16 to float32
//     __hip_bfloat16* o_local_ptr = o_ptr + pid_q_j * stride_on;
//     for (int d = 0; d < BLOCK_SIZE_D; ++d) {
//         if (d * stride_od < total_len * stride_on) { // Boundary check
//             acc_o[d] = __bfloat162float(o_local_ptr[d * stride_od]);
//         } else {
//             acc_o[d] = 0.0f;
//         }
//     }

//     float lse = lse_ptr[pid_q_j * stride_lse_n];
//     const float m_ij_last = m_ij_last_ptr[pid_q_j];

//     // ===================================================================================
//     // MAIN REDUCTION LOOP
//     // ===================================================================================
//     for (int block_id = 0; block_id < TOPK; ++block_id) {
//         const int t = t_shared[tid * TOPK + block_id];

//         if (t != -1) {
//             // Triton's branching logic translated to CUDA
//             int real_block_pos;
//             const float* l_ij_ptr;
//             const __hip_bfloat16* o_tiles_ptr;
//             const float* acc_o_scales_ptr;
//             int64_t stride_l_ij_b, stride_l_ij_n;
//             int64_t stride_otb, stride_otn;
//             int64_t stride_acc_b, stride_acc_n;

//             if (t == 0) {
//                 real_block_pos = 0;
//                 l_ij_ptr = l_ij_first_ptr;
//                 o_tiles_ptr = o_tiles_first_ptr;
//                 acc_o_scales_ptr = acc_o_scales_first_ptr;
//                 stride_l_ij_b = stride_l_ij_fb; stride_l_ij_n = stride_l_ij_fn;
//                 stride_otb = stride_otfb; stride_otn = stride_otfn;
//                 stride_acc_b = stride_acc_fb; stride_acc_n = stride_acc_fn;
//             } else {
//                 real_block_pos = t - 1;
//                 l_ij_ptr = l_ij_rest_ptr;
//                 o_tiles_ptr = o_tiles_rest_ptr;
//                 acc_o_scales_ptr = acc_o_scales_rest_ptr;
//                 stride_l_ij_b = stride_l_ij_rb; stride_l_ij_n = stride_l_ij_rn;
//                 stride_otb = stride_otrb; stride_otn = stride_otrn;
//                 stride_acc_b = stride_acc_rb; stride_acc_n = stride_acc_rn;
//             }

//             const int real_token_index = token_index_mapping_ptr[t * stride_tim_b + pid_q_j * stride_tim_n];

//             // --- Scalar loads from global memory (potential future optimization) ---
//             const float m_ij = m_ij_ptr[t * stride_m_ij_b + pid_q_j * stride_m_ij_n];
//             const float l_ij = l_ij_ptr[real_block_pos * stride_l_ij_b + real_token_index * stride_l_ij_n];

//             // --- LSE Update ---
//             const float delta = lse - m_ij;
//             const float log_delta = exp2_fast(delta) + l_ij;
//             lse = m_ij + log2_fast(log_delta);

//             // --- Vector loads and 'o' update ---
//             const __hip_bfloat16* o_tiles_local_ptr = o_tiles_ptr + real_block_pos * stride_otb + real_token_index * stride_otn;
//             const float acc_o_scale_tile = acc_o_scales_ptr[real_block_pos * stride_acc_b + real_token_index * stride_acc_n];

//             // This loop loads o_tiles (bf16), converts to float, and updates acc_o (float)
//             for (int d = 0; d < BLOCK_SIZE_D; ++d) {
//                 float o_tile_val = __bfloat162float(o_tiles_local_ptr[d * stride_otfd]);
//                 acc_o[d] = o_tile_val + acc_o[d] * acc_o_scale_tile;
//             }
//         }
//     }

//     // ===================================================================================
//     // FINALIZATION: Apply final scale and store results
//     // ===================================================================================
//     const float final_scale = exp2_fast(m_ij_last - lse);

//     // Store final 'o', converting from float32 accumulator to bf16 storage
//     for (int d = 0; d < BLOCK_SIZE_D; ++d) {
//          if (d * stride_od < total_len * stride_on) { // Boundary check
//             acc_o[d] *= final_scale;
//             o_local_ptr[d * stride_od] = __float2bfloat16_rn(acc_o[d]);
//         }
//     }

//     // Store final 'lse'
//     lse_ptr[pid_q_j * stride_lse_n] = lse;
// }


// Host function to launch the CUDA kernel
void lse_reduce_kernel_launcher(
    torch::Tensor lse,
    torch::Tensor m_ij,
    torch::Tensor l_ij_first,
    torch::Tensor l_ij_rest,
    torch::Tensor m_ij_last,
    torch::Tensor o,
    torch::Tensor o_tiles_first,
    torch::Tensor o_tiles_rest,
    torch::Tensor acc_o_scales_first,
    torch::Tensor acc_o_scales_rest,
    torch::Tensor t,
    torch::Tensor token_index_mapping,
    int start_head_id,
    int total_len,
    int topk
) {
    // Ensure all tensors are on the same CUDA device
    const auto device = t.device();
    TORCH_CHECK(lse.device() == device, "All tensors must be on the same device");
    TORCH_CHECK(o.device() == device, "All tensors must be on the same device");

    // Validate data types
    TORCH_CHECK(o.scalar_type() == torch::kBFloat16, "o must be bfloat16");
    TORCH_CHECK(o_tiles_first.scalar_type() == torch::kBFloat16, "o_tiles_first must be bfloat16");
    TORCH_CHECK(o_tiles_rest.scalar_type() == torch::kBFloat16, "o_tiles_rest must be bfloat16");
    TORCH_CHECK(lse.scalar_type() == torch::kFloat32, "lse must be float32");
    TORCH_CHECK(t.scalar_type() == torch::kInt32, "t must be int32");

    // Get problem dimensions
    const int num_heads = t.size(0);
    const int grid_z = num_heads; // One grid dimension for heads is common, but your triton code doesn't use it.
                                 // Let's stick to the Triton launch grid for now.

    const int num_qz_loop = total_len; // This seems to be the full length
    const dim3 grid( (total_len + WARP_SIZE - 1) / WARP_SIZE, 1, 1);
    const dim3 block(WARP_SIZE, 1, 1);

    // In your host-side code (before launching the kernel):

    // BLOCK_SIZE_D and TOPK are template parameters, so they'll be known at compile time
    // or passed as arguments to a wrapper function.
    // Let's assume BLOCK_SIZE_D and TOPK are available here.


    const int BLOCK_SIZE_D = o.size(2); // Assuming o is N x H x D
    TORCH_CHECK(BLOCK_SIZE_D == 64 || BLOCK_SIZE_D == 128, "Unsupported BLOCK_SIZE_D");


    // Calculate the offsets exactly as in the kernel
    size_t offset_t_shared = 0;
    size_t offset_real_token_index_shared = offset_t_shared + sizeof(int) * WARP_SIZE * (topk + SMEM_PAD_INT_OR_FLOAT);

    size_t offset_m_ij_stage0 = offset_real_token_index_shared + sizeof(int) * WARP_SIZE * (topk + SMEM_PAD_INT_OR_FLOAT);
    size_t offset_m_ij_stage1 = offset_m_ij_stage0 + sizeof(float) * WARP_SIZE;

    size_t offset_l_ij_stage0 = offset_m_ij_stage1 + sizeof(float) * WARP_SIZE;
    size_t offset_l_ij_stage1 = offset_l_ij_stage0 + sizeof(float) * WARP_SIZE;

    // The total shared memory size is the offset of the *last element* + its size.
    // The last array starts at offset_acc_o_scales_stage1 and contains WARP_SIZE floats.
    size_t shared_mem_size = offset_l_ij_stage1 + sizeof(float) * WARP_SIZE;
    // --- BEGIN DEBUG PRINT ---
    // printf("[Host] Launching reduce_kernel_cuda with configuration:\n");
    // printf("       - total_len: %d, TOPK: %d, BLOCK_SIZE_D: %d\n", total_len, topk, BLOCK_SIZE_D);
    // printf("       - Grid: (%u, %u, %u), Block: (%u, %u, %u)\n", grid.x, grid.y, grid.z, block.x, block.y, block.z);
    // printf("       - Shared Memory per Block: %zu bytes\n", shared_mem_size);
    // --- END DEBUG PRINT ---
    // This is a simplified dispatch. You might need a more complex one if D and TOPK vary a lot.
    if (BLOCK_SIZE_D == 128) {
        if (topk == 16) {
             lse_reduce_kernel_cuda<128, 16><<<grid, block, shared_mem_size, at::cuda::getCurrentCUDAStream()>>>(
                lse.data_ptr<float>(), m_ij.data_ptr<float>(), l_ij_first.data_ptr<float>(), l_ij_rest.data_ptr<float>(),
                m_ij_last.data_ptr<float>(),
                t.data_ptr<int>(), token_index_mapping.data_ptr<int>(),
                start_head_id, total_len,
                lse.stride(1),
                m_ij.stride(1), m_ij.stride(2),
                l_ij_first.stride(1), l_ij_first.stride(2),
                l_ij_rest.stride(1), l_ij_rest.stride(2),
                t.stride(1), t.stride(2),
                token_index_mapping.stride(1), token_index_mapping.stride(2)
            );
        } // Add else-if for other TOPK values
    } // Add else-if for other BLOCK_SIZE_D values

    // Check for any errors during kernel launch
    C10_CUDA_KERNEL_LAUNCH_CHECK();
}


// // Host function to launch the CUDA kernel
// void o_reduce_kernel_launcher(
//     torch::Tensor lse,
//     torch::Tensor m_ij,
//     torch::Tensor l_ij_first,
//     torch::Tensor l_ij_rest,
//     torch::Tensor m_ij_last,
//     torch::Tensor o,
//     torch::Tensor o_tiles_first,
//     torch::Tensor o_tiles_rest,
//     torch::Tensor acc_o_scales_first,
//     torch::Tensor acc_o_scales_rest,
//     torch::Tensor t,
//     torch::Tensor token_index_mapping,
//     int start_head_id,
//     int total_len,
//     int topk
// ) {
//     // Ensure all tensors are on the same CUDA device
//     const auto device = t.device();
//     TORCH_CHECK(lse.device() == device, "All tensors must be on the same device");
//     TORCH_CHECK(o.device() == device, "All tensors must be on the same device");

//     // Validate data types
//     TORCH_CHECK(o.scalar_type() == torch::kBFloat16, "o must be bfloat16");
//     TORCH_CHECK(o_tiles_first.scalar_type() == torch::kBFloat16, "o_tiles_first must be bfloat16");
//     TORCH_CHECK(o_tiles_rest.scalar_type() == torch::kBFloat16, "o_tiles_rest must be bfloat16");
//     TORCH_CHECK(lse.scalar_type() == torch::kFloat32, "lse must be float32");
//     TORCH_CHECK(t.scalar_type() == torch::kInt32, "t must be int32");

//     // Get problem dimensions
//     const int num_heads = t.size(0);
//     const int grid_z = num_heads; // One grid dimension for heads is common, but your triton code doesn't use it.
//                                  // Let's stick to the Triton launch grid for now.

//     const int num_qz_loop = total_len; // This seems to be the full length
//     const dim3 grid( (total_len + WARP_SIZE - 1) / WARP_SIZE, 1, 1);
//     const dim3 block(WARP_SIZE, 1, 1);

//     // In your host-side code (before launching the kernel):

//     // BLOCK_SIZE_D and TOPK are template parameters, so they'll be known at compile time
//     // or passed as arguments to a wrapper function.
//     // Let's assume BLOCK_SIZE_D and TOPK are available here.


//     const int BLOCK_SIZE_D = o.size(2); // Assuming o is N x H x D
//     TORCH_CHECK(BLOCK_SIZE_D == 64 || BLOCK_SIZE_D == 128, "Unsupported BLOCK_SIZE_D");


//     // Calculate the offsets exactly as in the kernel
//     size_t offset_t_shared = 0;
//     size_t offset_real_token_index_shared = offset_t_shared + sizeof(int) * WARP_SIZE * (topk + SMEM_PAD_INT_OR_FLOAT);

//     size_t offset_m_ij_stage0 = offset_real_token_index_shared + sizeof(int) * WARP_SIZE * (topk + SMEM_PAD_INT_OR_FLOAT);
//     size_t offset_m_ij_stage1 = offset_m_ij_stage0 + sizeof(float) * WARP_SIZE;

//     size_t offset_l_ij_stage0 = offset_m_ij_stage1 + sizeof(float) * WARP_SIZE;
//     size_t offset_l_ij_stage1 = offset_l_ij_stage0 + sizeof(float) * WARP_SIZE;

//     // The total shared memory size is the offset of the *last element* + its size.
//     // The last array starts at offset_acc_o_scales_stage1 and contains WARP_SIZE floats.
//     size_t shared_mem_size = offset_l_ij_stage1 + sizeof(float) * WARP_SIZE;
//     // --- BEGIN DEBUG PRINT ---
//     // printf("[Host] Launching reduce_kernel_cuda with configuration:\n");
//     // printf("       - total_len: %d, TOPK: %d, BLOCK_SIZE_D: %d\n", total_len, topk, BLOCK_SIZE_D);
//     // printf("       - Grid: (%u, %u, %u), Block: (%u, %u, %u)\n", grid.x, grid.y, grid.z, block.x, block.y, block.z);
//     // printf("       - Shared Memory per Block: %zu bytes\n", shared_mem_size);
//     // --- END DEBUG PRINT ---
//     // This is a simplified dispatch. You might need a more complex one if D and TOPK vary a lot.
//     if (BLOCK_SIZE_D == 128) {
//         if (topk == 16) {
//              o_reduce_kernel_cuda<128, 16><<<grid, block, shared_mem_size, at::cuda::getCurrentCUDAStream()>>>(
//                 lse.data_ptr<float>(),
//                 m_ij_last.data_ptr<float>(),
//                 reinterpret_cast<__hip_bfloat16*>(o.data_ptr()),
//                 reinterpret_cast<const __hip_bfloat16*>(o_tiles_first.data_ptr()),
//                 reinterpret_cast<const __hip_bfloat16*>(o_tiles_rest.data_ptr()),
//                 acc_o_scales_first.data_ptr<float>(), acc_o_scales_rest.data_ptr<float>(),
//                 t.data_ptr<int>(), token_index_mapping.data_ptr<int>(),
//                 start_head_id, total_len,
//                 lse.stride(1),
//                 o.stride(0), o.stride(2),
//                 o_tiles_first.stride(1), o_tiles_first.stride(2), o_tiles_first.stride(3),
//                 o_tiles_rest.stride(1), o_tiles_rest.stride(2), o_tiles_rest.stride(3),
//                 acc_o_scales_first.stride(1), acc_o_scales_first.stride(2),
//                 acc_o_scales_rest.stride(1), acc_o_scales_rest.stride(2),
//                 t.stride(1), t.stride(2),
//                 token_index_mapping.stride(1), token_index_mapping.stride(2)
//             );
//         } // Add else-if for other TOPK values
//     } // Add else-if for other BLOCK_SIZE_D values

//     // Check for any errors during kernel launch
//     C10_CUDA_KERNEL_LAUNCH_CHECK();
// }

// Host function to launch the CUDA kernel
void o_reduce_kernel_launcher(
    torch::Tensor lse,
    torch::Tensor m_ij,
    torch::Tensor l_ij_first,
    torch::Tensor l_ij_rest,
    torch::Tensor m_ij_last,
    torch::Tensor o,
    torch::Tensor o_tiles_first,
    torch::Tensor o_tiles_rest,
    torch::Tensor acc_o_scales_first,
    torch::Tensor acc_o_scales_rest,
    torch::Tensor t,
    torch::Tensor token_index_mapping,
    int start_head_id,
    int total_len,
    int topk
) {
    // Ensure all tensors are on the same CUDA device
    const auto device = t.device();
    TORCH_CHECK(lse.device() == device, "All tensors must be on the same device");
    TORCH_CHECK(o.device() == device, "All tensors must be on the same device");

    // Validate data types
    TORCH_CHECK(o.scalar_type() == torch::kBFloat16, "o must be bfloat16");
    TORCH_CHECK(o_tiles_first.scalar_type() == torch::kBFloat16, "o_tiles_first must be bfloat16");
    TORCH_CHECK(o_tiles_rest.scalar_type() == torch::kBFloat16, "o_tiles_rest must be bfloat16");
    TORCH_CHECK(lse.scalar_type() == torch::kFloat32, "lse must be float32");
    TORCH_CHECK(t.scalar_type() == torch::kInt32, "t must be int32");

    // Fixed template parameters
    constexpr int BLOCK_SIZE_D = 128;
    constexpr int TOPK = 16;
    constexpr int THREADS_PER_TOKEN = 32;
    constexpr int TOKENS_PER_BLOCK = 4;
    constexpr int THREADS_PER_BLOCK = 128;

    // Validate topk
    TORCH_CHECK(topk == TOPK, "topk must be ", TOPK);

    // Grid and block configuration
    const int num_blocks = (total_len + TOKENS_PER_BLOCK - 1) / TOKENS_PER_BLOCK;
    const dim3 grid(num_blocks, 1, 1);
    const dim3 block(THREADS_PER_BLOCK, 1, 1);

    // Calculate shared memory size based on kernel implementation
    size_t shared_mem_size = 0;
    
    // Static data for all tokens
    shared_mem_size += sizeof(int) * TOKENS_PER_BLOCK * TOPK; // t_shared
    shared_mem_size += sizeof(int) * TOKENS_PER_BLOCK * TOPK; // real_token_index_shared
    
    // Double buffered data - Stage 0
    shared_mem_size += sizeof(float) * TOKENS_PER_BLOCK; // acc_o_scales_stage0
    shared_mem_size += sizeof(float) * TOKENS_PER_BLOCK * BLOCK_SIZE_D; // o_tiles_stage0 (as float)
    
    // Double buffered data - Stage 1
    shared_mem_size += sizeof(float) * TOKENS_PER_BLOCK; // acc_o_scales_stage1
    shared_mem_size += sizeof(float) * TOKENS_PER_BLOCK * BLOCK_SIZE_D; // o_tiles_stage1 (as float)
    
    // Additional shared memory for constants per token
    shared_mem_size += sizeof(float) * TOKENS_PER_BLOCK; // shared_lse
    shared_mem_size += sizeof(float) * TOKENS_PER_BLOCK; // shared_m_ij_last
    shared_mem_size += sizeof(float) * TOKENS_PER_BLOCK; // shared_final_scale

    // Set CUDA device
    const at::cuda::CUDAGuard device_guard(device);

    // Launch kernel with fixed template parameters
    o_reduce_kernel_cuda<BLOCK_SIZE_D, TOPK><<<grid, block, shared_mem_size, at::cuda::getCurrentCUDAStream()>>>(
        // Pointers
        lse.data_ptr<float>(),
        m_ij_last.data_ptr<float>(),
        reinterpret_cast<__hip_bfloat16*>(o.data_ptr<at::BFloat16>()),
        reinterpret_cast<const __hip_bfloat16*>(o_tiles_first.data_ptr<at::BFloat16>()),
        reinterpret_cast<const __hip_bfloat16*>(o_tiles_rest.data_ptr<at::BFloat16>()),
        acc_o_scales_first.data_ptr<float>(),
        acc_o_scales_rest.data_ptr<float>(),
        t.data_ptr<int>(),
        token_index_mapping.data_ptr<int>(),
        // Scalars
        start_head_id,
        total_len,
        // Strides
        lse.stride(0),                    // stride_lse_n
        o.stride(0), o.stride(1),         // stride_on, stride_od
        o_tiles_first.stride(0), o_tiles_first.stride(1), o_tiles_first.stride(2), // stride_otfb, stride_otfn, stride_otfd
        o_tiles_rest.stride(0), o_tiles_rest.stride(1), o_tiles_rest.stride(2),    // stride_otrb, stride_otrn, stride_otrd
        acc_o_scales_first.stride(0), acc_o_scales_first.stride(1),                // stride_acc_fb, stride_acc_fn
        acc_o_scales_rest.stride(0), acc_o_scales_rest.stride(1),                  // stride_acc_rb, stride_acc_rn
        t.stride(0), t.stride(1),                                                  // stride_tn, stride_tk
        token_index_mapping.stride(0), token_index_mapping.stride(1)               // stride_tim_b, stride_tim_n
    );

    // Check for kernel launch errors
    C10_CUDA_KERNEL_LAUNCH_CHECK();
    
    // Optional: Check for kernel execution errors
    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel execution failed: ", hipGetErrorString(err));
}
